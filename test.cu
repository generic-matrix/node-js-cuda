
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
 
 //!!nvcc -c  test.cu --compiler-options -fPIC
 //!g++ -o program -L/usr/local/cuda/lib64 main.cpp test.o -lcuda -lcudart 
 
__global__ void add(float *a,float *b,float *c)
{
         *c = *a +*b;
}  
 
__global__ void sub(float *a,float *b,float *c)
{
         *c = *a -*b;
}  
 
__global__ void exp(float *a,float *c)
{
         *c = expf(*a);
}  
 
__global__ void mul(float *a,float *b,float *c)
{
         *c = *a * *b;
}  
 
__global__ void pow(float *a,float *b,float *c)
{
         *c = powf(*a,*b);
}  
 
 

float vecAdd_wrapper(float a,float b)
{
        float c;
        float*d_a, *d_b, *d_c; 
        int size = sizeof(float);
        
        hipMallocManaged((void **)&d_a, size);
        hipMallocManaged((void **)&d_b, size);
        hipMallocManaged((void **)&d_c, size);

        hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

        add<<<1,1>>>(d_a, d_b, d_c);

        hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        
        return c;
}

float vecSub_wrapper(float a,float b)
{
        float c;
        float*d_a, *d_b, *d_c; 
        int size = sizeof(float);
        
        hipMallocManaged((void **)&d_a, size);
        hipMallocManaged((void **)&d_b, size);
        hipMallocManaged((void **)&d_c, size);

        hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

        sub<<<1,1>>>(d_a, d_b, d_c);

        hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        
        return c;
}

float vecMul_wrapper(float a,float b)
{
        float c;
        float*d_a, *d_b, *d_c; 
        int size = sizeof(float);
        
        hipMallocManaged((void **)&d_a, size);
        hipMallocManaged((void **)&d_b, size);
        hipMallocManaged((void **)&d_c, size);

        hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

        mul<<<1,1>>>(d_a, d_b, d_c);

        hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        
        return c;
}

float vecExp_wrapper(float a)
{
        float c;
        float*d_a,*d_c; 
        int size = sizeof(float);
        
        hipMallocManaged((void **)&d_a, size);
        hipMallocManaged((void **)&d_c, size);

        hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);

        exp<<<1,1>>>(d_a, d_c);

        hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

        hipFree(d_a); hipFree(d_c);
        
        return c;
}


float vecPow_wrapper(float a,float b)
{
        float c;
        float*d_a, *d_b, *d_c; 
        int size = sizeof(float);
        
        hipMallocManaged((void **)&d_a, size);
        hipMallocManaged((void **)&d_b, size);
        hipMallocManaged((void **)&d_c, size);

        hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

        pow<<<1,1>>>(d_a, d_b, d_c);

        hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        
        return c;
}